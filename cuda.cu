#include "hip/hip_runtime.h"
/*
 * lab3 CAD 2021/2022 FCT/UNL
 * vad
 */
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "vsize.h"


/* read_ppm - read a PPM image ascii file
 *   returns pointer to data, dimensions and max colors (from PPM header)
 *   data format: sequence of width x height of 3 ints for R,G and B
 *   aborts on errors
 */
void read_ppm(FILE *f, int **img, int *width, int *height, int *maxcolors) {
    int count=0;
    char ppm[10];
    int c;
    // header
    while ( (c = fgetc(f))!=EOF && count<4 ) {
        if (isspace(c)) continue;
        if (c=='#') {
            while (fgetc(f) != '\n')
                ;
            continue;
        }
        ungetc(c,f);
        switch (count) {
            case 0: count += fscanf(f, "%2s", ppm); break;
            case 1: count += fscanf(f, "%d%d%d", width, height, maxcolors); break;
            case 2: count += fscanf(f, "%d%d", height, maxcolors); break;
            case 3: count += fscanf(f, "%d", maxcolors);
        }
    }
    assert(c!=EOF);
    assert(strcmp("P3", ppm)==0);
    // data
    int *data= *img = (int*)malloc(3*(*width)*(*height)*sizeof(int));
    assert(img!=NULL);
    int r,g,b, pos=0;
    while ( fscanf(f,"%d%d%d", &r, &g, &b)==3) {
        data[pos++] = r;
        data[pos++] = g;
        data[pos++] = b;
    }
    assert(pos==3*(*width)*(*height));
}


/* write_ppm - write a PPM image ascii file
 */
void write_ppm(FILE *f, int *img, int width, int height, int maxcolors) {
    // header
    fprintf(f, "P3\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++) {
        for (int c = 0; c < width; c++) {
            int p = 3 * (l * width + c);
            fprintf(f, "%d %d %d  ", img[p], img[p + 1], img[p + 2]);
        }
        fputc('\n',f);
    }
}


/* printImg - print to screen the content of img
 */
void printImg(int imgh, int imgw, const int *img) {
    for (int j=0; j < imgh; j++) {
        for (int i=0; i<imgw; i++) {
            int x = 3*(i+j*imgw);
            printf("%d,%d,%d  ", img[x], img[x+1], img[x+2]);
        }
        putchar('\n');
    }
}

__global__ void averageImg(int*out, int*img, int width, int height) {
    int line = blockIdx.x*blockDim.x+threadIdx.x;
    int col = blockIdx.y*blockDim.y+threadIdx.y;

    int r=0,g=0,b=0, n=0;
    for (int l=line-1; l<line+2 && l<height; l++)
        for (int c=col-1; c<col+2 && c<width; c++)
            if (l>=0 && c>=0) {
                int idx = 3*(l*width+c);
                r+=img[idx]; g+=img[idx+1]; b+=img[idx+2];
                n++;
            }
    int idx = 3*(line*width+col);
    out[idx]=r/n;
    out[idx+1]=g/n;
    out[idx+2]=b/n;
}


int main(int argc, char *argv[]) {
    int imgh, imgw, imgc;
    int *img;
    if (argc!=2) {
        fprintf(stderr,"usage: %s img.ppm\n", argv[0]);
        return EXIT_FAILURE;
    }
	FILE *f=fopen(argv[1],"r");
    if (f==NULL) {
        fprintf(stderr,"can't read file %s\n", argv[1]);
        return EXIT_FAILURE;
    }

    read_ppm(f, &img, &imgw, &imgh, &imgc);
	printf("PPM image %dx%dx%d\n", imgw, imgh, imgc);
//    printImg(imgh, imgw, img);

    dim3 dimBlock(NTHREADS, NTHREADS);
    dim3 dimGrid((imgw+dimBlock.x-1)/dimBlock.x, (imgh+dimBlock.y-1)/dimBlock.y);

    int *out = (int*)malloc(3*imgw*imgh*sizeof(int));
    assert(out!=NULL);

    int *img_cuda;
    int *out_cuda;
    hipMalloc(&img_cuda, 3*imgw*imgh*sizeof(int));
    hipMalloc(&out_cuda, 3*imgw*imgh*sizeof(int));
    if ( img_cuda==NULL || out_cuda==NULL ) {
        fprintf(stderr,"No GPU mem!\n");
        return EXIT_FAILURE;
    }
    hipMemcpy(img_cuda, img, 3*imgw*imgh*sizeof(int), hipMemcpyHostToDevice);

    clock_t t = clock();

    averageImg<<<dimGrid, dimBlock>>>(out_cuda, img_cuda, imgw, imgh);

    t = clock()-t;
    printf("time %f ms\n", t/(double)(CLOCKS_PER_SEC/1000));

    hipMemcpy(out, out_cuda, 3*imgh*imgw*sizeof(int), hipMemcpyDeviceToHost);

    //printImg(imgh, imgw, out);
    FILE *g=fopen("out_cuda.ppm", "w");
    write_ppm(g, out, imgw, imgh, imgc);
    fclose(g);
    return EXIT_SUCCESS;
}
